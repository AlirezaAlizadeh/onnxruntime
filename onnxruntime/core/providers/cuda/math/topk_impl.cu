#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "topk_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "hipcub/hipcub.hpp"
#include <limits>
#include <stdio.h>

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void FillInput(const T* input_x, T* output_v, int64_t* output_i, const int64_t* elem_nums, size_t size,
                          int64_t axis, int64_t K, int64_t offset, int64_t dimension) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, dimension);
  // const int64_t elem_nums_const[2] = {16 * 128, 128};

  int64_t left = offset / (axis == size - 1 ? 1 : elem_nums[axis + 1]) * elem_nums[axis];
  int64_t right = axis == size - 1 ? 0 : offset % elem_nums[axis + 1];
  int64_t input_offset = left + id * (axis == size - 1 ? 1 : elem_nums[axis + 1]) + right;

  printf(
      "input_x=%p output_v=%p output_i=%p elem_nums[0]=%lld elem_nums[1]=%lld "
      "id=%ld size=%lld axis=%lld k=%lld offset=%lld dimension=%lld "
      "left=%lld right=%lld input_offset=%lld\n",
      (void*)input_x, (void*)output_v, (void*)output_i, elem_nums[0], elem_nums[1], id, size, axis, K, offset, dimension,
      left, right, input_offset);

  output_v[id] = input_x[input_offset];
  output_i[id] = id;
}

template <typename T>
__global__ void FillOutput(const T* input_v, const int64_t* input_i, T* output_v, int64_t* output_i,
                           const int64_t* elem_nums, size_t size, int64_t axis, int64_t K, int64_t offset,
                           int64_t dimension) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, dimension);
  const int64_t elem_nums_const[2] = {16 * 128, 128};

  auto left = offset / (axis == size - 1 ? 1 : elem_nums_const[axis + 1]) * elem_nums_const[axis] * K / dimension;
  auto right = axis == size - 1 ? 0 : offset % elem_nums_const[axis + 1];
  auto output_offset = left + id * (axis == size - 1 ? 1 : elem_nums_const[axis + 1]) + right;

  printf(
      "input_v=%p input_i=%p output_v=%p output_i=%p elem_nums[0]=%lld elem_nums[1]=%lld "
      "id=%ld size=%lld axis=%lld k=%lld offset=%lld dimension=%lld\n"
      "left=%lld right=%lld output_offset=%lld\n",
      (void*)input_v, (void*)input_i, (void*)output_v, (void*)output_i, elem_nums[0], elem_nums[1],
      id, size, axis, K, offset, dimension,
      left, right, output_offset);

  output_v[output_offset] = input_v[id];
  output_i[output_offset] = input_i[id];
}

__global__ void ExcludeOutput(int64_t* output_i, int64_t K, int64_t dimension) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, dimension);
  if (id >= K) {
    output_i[id] = dimension;
  }
}

template <typename T>
Status TopKImpl(const CudaKernel* kernel, const T* input_x, T* output_v, int64_t* output_i, const int64_t* elem_nums,
                size_t size, int64_t axis, int64_t K, int64_t largest, int64_t sorted, int64_t N, int64_t dimension) {
  auto input_key_buffer = kernel->GetScratchBuffer<T>(dimension);  // the 'key' to sort on is the value in the input
  auto output_key_buffer = kernel->GetScratchBuffer<T>(dimension);
  auto input_value_buffer = kernel->GetScratchBuffer<int64_t>(dimension);  // the 'value' to store is the index of the value in the input
  auto output_value_buffer = kernel->GetScratchBuffer<int64_t>(dimension);
  auto input_key = input_key_buffer.get();
  auto output_key = output_key_buffer.get();
  auto input_value = input_value_buffer.get();
  auto output_value = output_value_buffer.get();
  size_t temp_bytes = 0;
  CUDA_RETURN_IF_ERROR(hipcub::DeviceRadixSort::SortPairs(nullptr, temp_bytes, input_key, output_key,
                                                       input_value, output_value, dimension,
                                                       0, sizeof(T) * 8, 0, /*debug*/ true));
  auto temp_storage_buffer = kernel->GetScratchBuffer<char>(temp_bytes);
  auto temp_storage = temp_storage_buffer.get();
  auto blocksPerGridD = (int)(ceil(static_cast<float>(dimension) / GridDim::maxThreadsPerBlock));
  auto blocksPerGridK = (int)(ceil(static_cast<float>(K) / GridDim::maxThreadsPerBlock));
  std::cout << "input_x:" << input_x << " elem_nums:" << (void*)elem_nums << "\n";

  for (int64_t i = 0; i < N; i++) {
    printf("i=%ld elem_nums=%p\n", i, (void*)elem_nums);
    CUDA_CALL_THROW(hipDeviceSynchronize());
    printf("FillInput\n");
    //int64_t input_offset = left + id * (axis == size - 1 ? 1 : elem_nums_cpu[axis + 1]) + right;
    //printf("left=%lld right=%lld input_offset=%lld\n", left, right, input_offset);

    FillInput<T><<<blocksPerGridD, GridDim::maxThreadsPerBlock, 0>>>(input_x, input_key, input_value, elem_nums,
                                                                     size, axis, K, i, dimension);
    CUDA_CALL_THROW(hipDeviceSynchronize());

    printf("Sort\n");
    CUDA_RETURN_IF_ERROR(1 == largest
                             ? hipcub::DeviceRadixSort::SortPairsDescending(temp_storage, temp_bytes, input_key, output_key,
                                                                         input_value, output_value, dimension)
                             : hipcub::DeviceRadixSort::SortPairs(temp_storage, temp_bytes, input_key, output_key,
                                                               input_value, output_value, dimension));
    printf("Post sort largest=%ld\n", largest);
    CUDA_CALL_THROW(hipDeviceSynchronize());
    if (1 == sorted) {
      printf("FillOutput\n");
      FillOutput<T><<<blocksPerGridK, GridDim::maxThreadsPerBlock, 0>>>(output_key, output_value, output_v, output_i,
                                                                        elem_nums, size, axis, K, i, dimension);
      CUDA_CALL_THROW(hipDeviceSynchronize());
    } else {  //reorder by ascending index
      std::cout << "ExcludeOutput\n";
      ExcludeOutput<<<blocksPerGridD, GridDim::maxThreadsPerBlock, 0>>>(output_value, K, dimension);
      CUDA_RETURN_IF_ERROR(hipcub::DeviceRadixSort::SortPairs(temp_storage, temp_bytes, output_value, input_value, output_key, input_key, dimension));
      FillOutput<T><<<blocksPerGridK, GridDim::maxThreadsPerBlock, 0>>>(input_key, input_value, output_v, output_i, elem_nums, size, axis, K, i, dimension);
      CUDA_CALL_THROW(hipDeviceSynchronize());
    }
  }
  return Status::OK();
}

#define TOPKIMPLE(T) template Status TopKImpl<T>(const CudaKernel* kernel, \
                                                 const T* input_x,         \
                                                 T* output_v,              \
                                                 int64_t* output_i,        \
                                                 const int64_t* elem_nums, \
                                                 size_t size,              \
                                                 int64_t axis,             \
                                                 int64_t K,                \
                                                 int64_t largest,          \
                                                 int64_t sorted,           \
                                                 int64_t N,                \
                                                 int64_t dimension)

TOPKIMPLE(uint8_t);
TOPKIMPLE(uint16_t);
TOPKIMPLE(uint32_t);
TOPKIMPLE(uint64_t);
TOPKIMPLE(int8_t);
TOPKIMPLE(int16_t);
TOPKIMPLE(int32_t);
TOPKIMPLE(int64_t);
TOPKIMPLE(float);
TOPKIMPLE(double);

}  // namespace cuda
}  // namespace onnxruntime
